#include "hip/hip_runtime.h"
//=============================================================================================================
/**
* @file     rapmusic_cuda.cu
* @author   Christoph Dinh <christoph.dinh@live.de>;
* @version  1.0
* @date     March, 2011
*
* @section  LICENSE
*
* Copyright (C) 2011 Christoph Dinh. All rights reserved.
*
* No part of this program may be photocopied, reproduced,
* or translated to another program language without the
* prior written consent of the author.
*
*
* @brief    ToDo Documentation...
*
*/


//*************************************************************************************************************
//=============================================================================================================
// CUDA INCLUDES
//=============================================================================================================

#include "../include/rapmusic_cuda.cuh"

#include "../include/cudadevice.cuh"
#include "../include/handle_error.cuh"
#include "../include/rapmusic_kernel.cuh"

#include "../include/cuhpcvalue.cuh"


//*************************************************************************************************************
//=============================================================================================================
// CPP INCLUDES
//=============================================================================================================


#include "../../cpp/include/eigeninterface.h"

#include "../../cpp/include/model.h"

#include "../../cpp/include/rapdipoles.h"


//*************************************************************************************************************
//=============================================================================================================
// DEFINE NAMESPACE HPCLib
//=============================================================================================================

namespace HPCLib
{

//*************************************************************************************************************
//=============================================================================================================
// USED NAMESPACES
//=============================================================================================================


//*************************************************************************************************************
//=============================================================================================================
// DEFINE MEMBER METHODS
//=============================================================================================================

RapMusic_Cuda::RapMusic_Cuda()
: m_iPairCols(6)
, m_iMaxBlocksPerMultiProcessor(8) //CUDA C Programming Guide - Appendix F
{

}


//*************************************************************************************************************

RapMusic_Cuda::~RapMusic_Cuda()
{
    m_host_pLeadFieldMat = NULL;

    //garbage collecting
    //######## CUDA START ########
        // free the memory allocated on the GPU
        /*HANDLE_ERROR( hipFree( m_dev_pLeadFieldMat ) );*/
        delete m_dev_pLeadFieldMat;

        delete m_dev_pVecPairIdxCombinations;
    //######## CUDA END ########
}


//*************************************************************************************************************

//template <class T>
bool RapMusic_Cuda::initRAPMusic(   HPCLib::CudaDevice* p_pDeviceInfo,
                                    HPCLib::Model<float>* p_pModel,
                                    bool p_bSparsed, int p_iN, double p_dThr)
{
    return initRAPMusic(p_pDeviceInfo,
                        p_bSparsed ? p_pModel->getSparsedLeadFieldMat() : p_pModel->getLeadFieldMat(),
                        p_bSparsed ? p_pModel->getSparsedGridMat() : p_pModel->getGridMat(),
                        p_iN, p_dThr);
}


//*************************************************************************************************************

//template <class T>
bool RapMusic_Cuda::initRAPMusic(   HPCLib::CudaDevice* p_pDeviceInfo,
                                    HPCMatrix<float>* p_pMatLeadField,
                                    HPCMatrix<float>* p_pMatGrid,
                                    int p_iN, double p_dThr)
{
    m_iMultiProcessorCount = p_pDeviceInfo->getSelectedDeviceProperties().multiProcessorCount;//14;
    m_iWarpSize = p_pDeviceInfo->getSelectedDeviceProperties().warpSize;//32;
    m_iMaxThreadsPerMultiProcessor =  p_pDeviceInfo->getSelectedDeviceProperties().maxThreadsPerMultiProcessor;//1536;
    m_iSharedMemoryPerMultiProcessor = p_pDeviceInfo->getSelectedDeviceProperties().sharedMemPerBlock;//48*1024;

    hipblasStatus_t status = cublasInit ();

    //Initialize RAP-MUSIC
    std::cout << "##### Initialization CUDA RAP MUSIC started ######\n\n";

    m_iN = p_iN;
    m_dThreshold = p_dThr;

    //Grid check
    if(p_pMatGrid != NULL)
    {
        if ( p_pMatGrid->rows() != p_pMatLeadField->cols() / 3 )
        {
            std::cout << "Grid does not fit to given Lead Field!\n";
            return false;
        }
    }

    m_pMatGrid = p_pMatGrid;

    //Lead Fiel check
    if ( p_pMatLeadField->cols() % 3 != 0 )
    {
        std::cout << "Lead Field is not associated with a 3D grid!\n";
        return false;
    }

    m_pMatLeadField = p_pMatLeadField;

    m_dev_pLeadFieldMat = new cuHPCMatrix<float>(*p_pMatLeadField);//### CUDA ###

    m_iNumGridPoints = (int)(m_dev_pLeadFieldMat->cols()/3);
    m_iNumChannels = m_dev_pLeadFieldMat->rows();

    //##### Calc lead field combination #####

    std::cout << "Calculate lead field combinations. \n";

    m_iNumLeadFieldCombinations = nchoose2(m_iNumGridPoints+1);

    //######## CUDA START ########
        // allocate device vector
        m_dev_pVecPairIdxCombinations = new thrust::device_vector<int>(2 * m_iNumLeadFieldCombinations);
        // obtain raw pointer to device vector�s memory -> for usage in kernel
        m_dev_pPairIdxCombinations = thrust::raw_pointer_cast(&(*m_dev_pVecPairIdxCombinations)[0]);

        cuCalcPairCombinations<<<128,1>>>( m_iNumGridPoints, m_iNumLeadFieldCombinations, m_dev_pPairIdxCombinations);
    //######## CUDA END ########

    std::cout << "Lead Field combinations calculated. \n\n";

    //##### Calc lead field combination end #####

    std::cout << "Number of grid points: " << m_iNumGridPoints << "\n\n";

    std::cout << "Number of combinated points: " << m_iNumLeadFieldCombinations << "\n\n";

    std::cout << "Number of sources to find: " << m_iN << "\n\n";

    std::cout << "Threshold: " << m_dThreshold << "\n\n";

    //Init end

    std::cout << "##### Initialization CUDA RAP MUSIC completed ######\n\n\n";

    m_bIsInit = true;

    return m_bIsInit;
}


//*************************************************************************************************************

bool RapMusic_Cuda::calcRapMusic(HPCMatrix<float>* p_pMatMeasurement, RapDipoles<float>*& p_pRapDipoles)
{
    //if not initialized -> break
    if(!m_bIsInit)
    {
        std::cout << "RAP-Music wasn't initialized!"; //ToDo: catch this earlier
        return false;
    }

    //Test if data are correct
    if(p_pMatMeasurement->rows() != m_iNumChannels)
    {
        std::cout << "Lead Field channels do not fit to number of measurement channels!"; //ToDo: catch this earlier
        return false;
    }

//     //Inits
//     //Stop the time for benchmark purpose
//     clock_t start, end;
//     start = clock();


    //Calculate the signal subspace (t_dev_pMatPhi_s)
    cuHPCMatrix<float>* t_dev_pMatPhi_s = NULL;//(m_iNumChannels, t_r < m_iN ? m_iN : t_r);
    //separate kernel for calcPhi_s -> not possible because measurement is often too big for shared memory
    int t_r = calcPhi_s(*p_pMatMeasurement, t_dev_pMatPhi_s);


    int t_iMaxSearch = m_iN < t_r ? m_iN : t_r; //The smallest of Rank and Iterations

    if (t_r < m_iN)
    {
        std::cout << "Warning: Rank " << t_r << " of the measurement data is smaller than the " << m_iN;
        std::cout << " sources to find." << std::endl;
        std::cout << "         Searching now for " << t_iMaxSearch << " correlated sources.";
        std::cout << std::endl << std::endl;
    }

    //Create Orthogonal Projector
    //OrthProj
    HPCMatrix<float> t_matOrthProj(m_iNumChannels,m_iNumChannels);
    t_matOrthProj.setIdentity();

    cuHPCMatrix<float>* t_dev_pMatOrthProj = new cuHPCMatrix<float>(t_matOrthProj);//### CUDA ###

    //A_k_1
    HPCMatrix<float> t_matA_k_1(m_iNumChannels,t_iMaxSearch);
    t_matA_k_1.reset(0.0);//setZero();

    if (m_pMatGrid != NULL)
    {
        if(p_pRapDipoles != NULL)
            p_pRapDipoles->initRapDipoles(m_pMatGrid);
        else
            p_pRapDipoles = new RapDipoles<float>(m_pMatGrid);
    }
    else
    {
        if(p_pRapDipoles != NULL)
            delete p_pRapDipoles;

        p_pRapDipoles = new RapDipoles<float>();
    }

    std::cout << "##### Calculation of CUDA RAP MUSIC started ######\n\n";


    cuHPCMatrix<float>* t_dev_pMatProj_Phi_s = new cuHPCMatrix<float>(t_matOrthProj.rows(), t_dev_pMatPhi_s->cols());//### CUDA ###



    //new Version: Calculate projection before
    HPCMatrix<float> t_matProj_LeadField(m_dev_pLeadFieldMat->rows(), m_dev_pLeadFieldMat->cols());

    cuHPCMatrix<float>* t_dev_pMatProj_LeadField = new cuHPCMatrix<float>(m_dev_pLeadFieldMat->rows(), m_dev_pLeadFieldMat->cols());//### CUDA ###


    for(int r = 0; r < t_iMaxSearch; ++r)
    {

        //ToDO needs to be checked whether using device pointer is performant
        //t_dev_pMatProj_Phi_s = t_dev_pMatOrthProj*t_dev_pMatPhi_s;
        t_dev_pMatProj_Phi_s->cuHPCMatMult('N', 'N',*t_dev_pMatOrthProj,*t_dev_pMatPhi_s);//### CUDA ###


        //new Version: Calculating Projection before -> ToDo remove this later on
        t_matProj_LeadField = t_matOrthProj * (*m_pMatLeadField);//Subtract the found sources from the current found source

        //t_dev_pMatProj_LeadField = t_dev_pMatOrthProj*m_dev_pLeadFieldMat;
        t_dev_pMatProj_LeadField->cuHPCMatMult('N', 'N',*t_dev_pMatOrthProj,*m_dev_pLeadFieldMat);//### CUDA ###


        //###First Option###
        //Step 1: lt. Mosher 1998 -> Maybe tmp_Proj_Phi_S is already orthogonal -> so no SVD needed -> U_B = tmp_Proj_Phi_S;

            cuHPCMatrix<float>* t_dev_pMatU_B = new cuHPCMatrix<float>(t_dev_pMatPhi_s->rows(), t_dev_pMatPhi_s->cols());//### CUDA ###

            cuHPCValue<int> t_dev_iRank(0);//### CUDA ###


            int t_iTh_y = 8;//16; //ToDo: More than 8 threads - wrong results
            int t_iTh_z = 2;//1;

            int t_iMatSize_U_B = t_dev_pMatPhi_s->rows() * t_dev_pMatPhi_s->cols();
            int t_iWMatSize_U_B = t_dev_pMatPhi_s->cols();
            int t_iCacheYZSize_U_B = t_iTh_y*t_iTh_z;
            int t_iSVDCache_U_B = t_dev_pMatPhi_s->cols()+1+1;//rv1[m_iPairCols]; scale; s

            size_t t_iSharedMem_U_B = sizeof(float) * (t_iMatSize_U_B + t_iWMatSize_U_B + t_iCacheYZSize_U_B + t_iSVDCache_U_B);

            dim3 blocks = dim3( 1, 1);
            dim3 threads = dim3( 1, t_iTh_y, t_iTh_z);



            cuCalcU_B<<< blocks, threads, t_iSharedMem_U_B >>>
                                 ( t_dev_pMatProj_Phi_s->data(),
                                   t_dev_pMatProj_Phi_s->rows(),
                                   t_dev_pMatProj_Phi_s->cols(),
                                   t_dev_pMatU_B->data(),
                                   t_dev_iRank.data());
            HANDLE_ERROR( hipDeviceSynchronize() ); //to ensure that the kernel has completed

            int t_iFullRank_U_B = t_dev_iRank.toHostValue();



            HPCMatrix<float> t_matU_B(t_dev_pMatProj_Phi_s->rows(), t_iFullRank_U_B);
            // copy the array back from the GPU to the CPU
            HANDLE_ERROR( hipMemcpy( t_matU_B.data(), t_dev_pMatU_B->data(),
                              t_iFullRank_U_B * t_dev_pMatProj_Phi_s->rows() * sizeof(float),
                              hipMemcpyDeviceToHost ) );

            //ToDo - better to resize - drop no longer needed columns
            delete t_dev_pMatU_B;
            t_dev_pMatU_B = new cuHPCMatrix<float>(t_matU_B);//### CUDA ###



        //######## CUDA START ########
            // allocate device vector
            thrust::device_vector<float> t_dev_vecRoh(m_iNumLeadFieldCombinations);
            // obtain raw pointer to device vector�s memory -> for usage in kernel
            float * t_dev_pRoh = thrust::raw_pointer_cast(&t_dev_vecRoh[0]);
        //######## CUDA END ########

// subcorr GPU

        //######## CUDA START ########
            hipEvent_t start, stop;
            float   elapsedTime;

            HANDLE_ERROR( hipEventCreate( &start ) );
            HANDLE_ERROR( hipEventCreate( &stop ) );


            t_iTh_y = 8;//16; //ToDo: More than 8 threads - wrong results
            t_iTh_z = 2;//1;

            int t_iPairMatSize = m_dev_pLeadFieldMat->rows() * m_iPairCols;
            int t_iWMatSize = m_iPairCols;
            int t_iCorMatSize = m_iPairCols*t_iFullRank_U_B;
            int t_iCacheYZSize = t_iTh_y*t_iTh_z;
            int t_iSVDCache = m_iPairCols+1+1;//rv1[m_iPairCols]; scale; s

            size_t t_iSharedMemPerPairMat = sizeof(float) * (t_iPairMatSize + t_iWMatSize + t_iCorMatSize + t_iCacheYZSize + t_iSVDCache);
            int t_iPairMatsPerMultiProcessor = m_iSharedMemoryPerMultiProcessor/t_iSharedMemPerPairMat;
            std::cout << "Shared Memory Usage: " << t_iSharedMemPerPairMat << " Byte x " << t_iPairMatsPerMultiProcessor << std::endl;

            int t_iPairMatsPerBlock = ceil((float)(t_iPairMatsPerMultiProcessor)/(float)m_iMaxBlocksPerMultiProcessor);//=threadDim.x
/*DIRTY HACK*/  t_iPairMatsPerBlock = 2;//t_iPairMatsPerBlock > 2 ? 2 : t_iPairMatsPerBlock;//ToDo Debug when 3 Mats per Block we get the wrong result
            std::cout << "Pair Mats per Block: " << t_iPairMatsPerBlock << std::endl;

            size_t t_iSharedMemPerBlock = t_iSharedMemPerPairMat * t_iPairMatsPerBlock;

            blocks = dim3( /*7381*/ 64*m_iMultiProcessorCount, 1);
            threads = dim3( t_iPairMatsPerBlock, t_iTh_y, t_iTh_z);


            HANDLE_ERROR( hipEventRecord( start, 0 ) );

            RapMusicSubcorr<<<blocks, threads, t_iSharedMemPerBlock>>>
                                              ( t_dev_pMatProj_LeadField->data(),
                                                t_dev_pMatProj_LeadField->rows(),
                                                t_dev_pMatProj_LeadField->cols(),
                                                m_dev_pPairIdxCombinations,
                                                m_iNumLeadFieldCombinations,

                                                t_dev_pMatU_B->data(),
                                                t_iFullRank_U_B,
                                                t_dev_pRoh );
            HANDLE_ERROR( hipDeviceSynchronize() ); //to ensure that the kernel has completed

            HANDLE_ERROR( hipEventRecord( stop, 0 ) );
            HANDLE_ERROR( hipEventSynchronize( stop ) );
            HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                                start, stop ) );

            // free events
            HANDLE_ERROR( hipEventDestroy( start ) );
            HANDLE_ERROR( hipEventDestroy( stop ) );

            std::cout << "Time Elapsed: " << elapsedTime << " ms" << std::endl;
        //######## CUDA END ########

        //Find the maximum of correlation 
        //######## CUDA THRUST START ########
            //max_element returns an iterator, so to convert that into a position we subtract the iterator at the beginning of the vector.
            int t_iMaxIdx = thrust::max_element(t_dev_vecRoh.begin(), t_dev_vecRoh.end()) - t_dev_vecRoh.begin(); 
            float t_val_roh_k = t_dev_vecRoh[t_iMaxIdx];
        //######## THRUST CUDA END ########

        //get positions in sparsed leadfield from index combinations;
        int t_iIdx1 = (*m_dev_pVecPairIdxCombinations)[2*t_iMaxIdx];
        int t_iIdx2 = (*m_dev_pVecPairIdxCombinations)[2*t_iMaxIdx+1];

        // (Idx+1) because of MATLAB positions -> starting with 1 not with 0
        std::cout << "Iteration: " << r+1 << " of " << t_iMaxSearch
            << "; Correlation: " << t_val_roh_k<< "; Position (Idx+1): " << t_iIdx1+1 << " - " << t_iIdx2+1 <<"\n\n";

        //Calculations with the max correlated dipole pair G_k_1
        HPCMatrix<float> t_matG_k_1(t_matProj_LeadField.rows(),6);
        getLeadFieldPair(*m_pMatLeadField, t_matG_k_1, t_iIdx1, t_iIdx2);

        HPCMatrix<float> t_matProj_G_k_1(t_matOrthProj.rows(), t_matG_k_1.cols());
        t_matProj_G_k_1 = t_matOrthProj * t_matG_k_1;//Subtract the found sources from the current found source

        //Calculate source direction
        //source direction (p_pMatPhi) for current source r (phi_k_1)
        HPCMatrix<float> t_vec_phi_k_1(6, 1);
        subcorr(t_matProj_G_k_1, t_matU_B, t_vec_phi_k_1);//Correlate the current source to calculate the direction

        //Set return values
        p_pRapDipoles->insertSource(t_iIdx1, t_iIdx2, t_vec_phi_k_1.data(), t_val_roh_k);
        
        //Stop Searching when Correlation is smaller then the Threshold
        if (t_val_roh_k < m_dThreshold)
        {
            std::cout << "Searching stopped, last correlation " << t_val_roh_k;
            std::cout << " is smaller then the given threshold " << m_dThreshold << std::endl << std::endl;
            break;
        }
        
        //Calculate A_k_1 = [a_theta_1..a_theta_k_1] matrix for subtraction of found source
        calcA_k_1(t_matG_k_1, t_vec_phi_k_1, r, t_matA_k_1);

        //Calculate new orthogonal Projector (Pi_k_1)
        calcOrthProj(t_matA_k_1, t_matOrthProj);


        //#### CUDA START ####
            HANDLE_ERROR( hipMemcpy(   t_dev_pMatOrthProj->data(),
                                        t_matOrthProj.data(),
                                        sizeof(float) * t_matOrthProj.size(),
                                        hipMemcpyHostToDevice ) );
        //#### CUDA END ####


        //garbage collecting
            // free the memory allocated on the GPU
            delete t_dev_pMatU_B;

            // free the memory we allocated on the CPU

    }

    //garbage collecting
        // free the memory allocated on the GPU
        delete t_dev_pMatProj_LeadField;

        delete t_dev_pMatProj_Phi_s;

        delete t_dev_pMatOrthProj;

        delete t_dev_pMatPhi_s;

        // free the memory we allocated on the CPU

    std::cout << "##### Calculation of CUDA RAP MUSIC completed ######\n\n";

//     end = clock();
// 
//     float t_fElapsedTime = ( (float)(end-start) / (float)CLOCKS_PER_SEC ) * 1000.0f;
//     std::cout << "Time Elapsed: " << t_fElapsedTime << " ms" << std::endl << std::endl;


    //garbage collecting
    //ToDo


    return true;
}


//*************************************************************************************************************

bool RapMusic_Cuda::calcPowellRAPMusic(HPCMatrix<float>* p_pMatMeasurement, RapDipoles<float>*& p_pRapDipoles)
{
    //if not initialized -> break
    if(!m_bIsInit)
    {
        std::cout << "RAP-Music wasn't initialized!"; //ToDo: catch this earlier
        return false;
    }

    //Test if data are correct
    if(p_pMatMeasurement->rows() != m_iNumChannels)
    {
        std::cout << "Lead Field channels do not fit to number of measurement channels!"; //ToDo: catch this earlier
        return false;
    }

//     //Inits
//     //Stop the time for benchmark purpose
//     clock_t start, end;
//     start = clock();


    //Calculate the signal subspace (t_dev_pMatPhi_s)
    cuHPCMatrix<float>* t_dev_pMatPhi_s = NULL;//(m_iNumChannels, t_r < m_iN ? m_iN : t_r);
    //separate kernel for calcPhi_s -> not possible because measurement is often too big for shared memory
    int t_r = calcPhi_s(*p_pMatMeasurement, t_dev_pMatPhi_s);


    int t_iMaxSearch = m_iN < t_r ? m_iN : t_r; //The smallest of Rank and Iterations

    if (t_r < m_iN)
    {
        std::cout << "Warning: Rank " << t_r << " of the measurement data is smaller than the " << m_iN;
        std::cout << " sources to find." << std::endl;
        std::cout << "         Searching now for " << t_iMaxSearch << " correlated sources.";
        std::cout << std::endl << std::endl;
    }

    //Create Orthogonal Projector
    //OrthProj
    HPCMatrix<float> t_matOrthProj(m_iNumChannels,m_iNumChannels);
    t_matOrthProj.setIdentity();

    cuHPCMatrix<float>* t_dev_pMatOrthProj = new cuHPCMatrix<float>(t_matOrthProj);//### CUDA ###

    //A_k_1
    HPCMatrix<float> t_matA_k_1(m_iNumChannels,t_iMaxSearch);
    t_matA_k_1.reset(0.0);//setZero();

    if (m_pMatGrid != NULL)
    {
        if(p_pRapDipoles != NULL)
            p_pRapDipoles->initRapDipoles(m_pMatGrid);
        else
            p_pRapDipoles = new RapDipoles<float>(m_pMatGrid);
    }
    else
    {
        if(p_pRapDipoles != NULL)
            delete p_pRapDipoles;

        p_pRapDipoles = new RapDipoles<float>();
    }

    std::cout << "##### Calculation of CUDA RAP MUSIC started ######\n\n";


    cuHPCMatrix<float>* t_dev_pMatProj_Phi_s = new cuHPCMatrix<float>(t_matOrthProj.rows(), t_dev_pMatPhi_s->cols());//### CUDA ###



    //new Version: Calculate projection before
    HPCMatrix<float> t_matProj_LeadField(m_dev_pLeadFieldMat->rows(), m_dev_pLeadFieldMat->cols());

    cuHPCMatrix<float>* t_dev_pMatProj_LeadField = new cuHPCMatrix<float>(m_dev_pLeadFieldMat->rows(), m_dev_pLeadFieldMat->cols());//### CUDA ###


    for(int r = 0; r < t_iMaxSearch; ++r)
    {

        //ToDO needs to be checked whether using device pointer is performant
        //t_dev_pMatProj_Phi_s = t_dev_pMatOrthProj*t_dev_pMatPhi_s;
        t_dev_pMatProj_Phi_s->cuHPCMatMult('N', 'N',*t_dev_pMatOrthProj,*t_dev_pMatPhi_s);//### CUDA ###


        //new Version: Calculating Projection before -> ToDo remove this later on
        t_matProj_LeadField = t_matOrthProj * (*m_pMatLeadField);//Subtract the found sources from the current found source

        //t_dev_pMatProj_LeadField = t_dev_pMatOrthProj*m_dev_pLeadFieldMat;
        t_dev_pMatProj_LeadField->cuHPCMatMult('N', 'N',*t_dev_pMatOrthProj,*m_dev_pLeadFieldMat);//### CUDA ###


        //###First Option###
        //Step 1: lt. Mosher 1998 -> Maybe tmp_Proj_Phi_S is already orthogonal -> so no SVD needed -> U_B = tmp_Proj_Phi_S;

            cuHPCMatrix<float>* t_dev_pMatU_B = new cuHPCMatrix<float>(t_dev_pMatPhi_s->rows(), t_dev_pMatPhi_s->cols());//### CUDA ###

            cuHPCValue<int> t_dev_iRank(0);//### CUDA ###


            int t_iTh_y = 8;//16; //ToDo: More than 8 threads - wrong results
            int t_iTh_z = 2;//1;

            int t_iMatSize_U_B = t_dev_pMatPhi_s->rows() * t_dev_pMatPhi_s->cols();
            int t_iWMatSize_U_B = t_dev_pMatPhi_s->cols();
            int t_iCacheYZSize_U_B = t_iTh_y*t_iTh_z;
            int t_iSVDCache_U_B = t_dev_pMatPhi_s->cols()+1+1;//rv1[m_iPairCols]; scale; s

            size_t t_iSharedMem_U_B = sizeof(float) * (t_iMatSize_U_B + t_iWMatSize_U_B + t_iCacheYZSize_U_B + t_iSVDCache_U_B);

            dim3 blocks = dim3( 1, 1);
            dim3 threads = dim3( 1, t_iTh_y, t_iTh_z);



            cuCalcU_B<<< blocks, threads, t_iSharedMem_U_B >>>
                                 ( t_dev_pMatProj_Phi_s->data(),
                                   t_dev_pMatProj_Phi_s->rows(),
                                   t_dev_pMatProj_Phi_s->cols(),
                                   t_dev_pMatU_B->data(),
                                   t_dev_iRank.data());
            HANDLE_ERROR( hipDeviceSynchronize() ); //to ensure that the kernel has completed

            int t_iFullRank_U_B = t_dev_iRank.toHostValue();



            HPCMatrix<float> t_matU_B(t_dev_pMatProj_Phi_s->rows(), t_iFullRank_U_B);
            // copy the array back from the GPU to the CPU
            HANDLE_ERROR( hipMemcpy( t_matU_B.data(), t_dev_pMatU_B->data(),
                              t_iFullRank_U_B * t_dev_pMatProj_Phi_s->rows() * sizeof(float),
                              hipMemcpyDeviceToHost ) );

            //ToDo - better to resize - drop no longer needed columns
            delete t_dev_pMatU_B;
            t_dev_pMatU_B = new cuHPCMatrix<float>(t_matU_B);//### CUDA ###



        //######## CUDA START ########
            // allocate device vector
            thrust::device_vector<float> t_dev_vecRoh(m_iNumLeadFieldCombinations);
            // obtain raw pointer to device vector�s memory -> for usage in kernel
            float * t_dev_pRoh = thrust::raw_pointer_cast(&t_dev_vecRoh[0]);

            // allocate device vector
            thrust::device_vector<int> t_dev_vecRowIndezes(m_iNumGridPoints);
            // obtain raw pointer to device vector�s memory -> for usage in kernel
            int * t_dev_pRowIndezes = thrust::raw_pointer_cast(&t_dev_vecRowIndezes[0]);

        //######## CUDA END ########

// subcorr GPU

        //Powell
        int t_iCurrentRow = 2;

        int t_iIdx1 = -1;
        int t_iIdx2 = -1;

        int t_iMaxIdx_old = -1;
        int t_iMaxIdx = -1;

        int t_iMaxFound = 0;

        float t_val_roh_k = 0;

        int t_iNumVecElements = m_iNumGridPoints;

        while(t_iMaxFound == 0)
        {


        //######## CUDA START ########
            hipEvent_t start, stop;
            float   elapsedTime;

            HANDLE_ERROR( hipEventCreate( &start ) );
            HANDLE_ERROR( hipEventCreate( &stop ) );

            //Powell Indizes
            cuPowellIdxVec<<<32, 32>>>( t_iCurrentRow,
                                        t_iNumVecElements,
                                        t_dev_pRowIndezes );
            HANDLE_ERROR( hipDeviceSynchronize() ); //to ensure that the kernel has completed

//             //DEBUG
//             thrust::host_vector<int> h_vec(m_iNumGridPoints);
//             // transfer data back to host
//             thrust::copy(t_dev_vecRowIndezes.begin(), t_dev_vecRowIndezes.end(), h_vec.begin());
//             std::cout << "indezes" << std::endl;
//              for(int i = 0; i < 10; ++i)
//                  std::cout << h_vec[i] << std::endl;
//              //DEBUG

            t_iTh_y = 8;//16; //ToDo: More than 8 threads - wrong results
            t_iTh_z = 2;//1;

            int t_iPairMatSize = m_dev_pLeadFieldMat->rows() * m_iPairCols;
            int t_iWMatSize = m_iPairCols;
            int t_iCorMatSize = m_iPairCols*t_iFullRank_U_B;
            int t_iCacheYZSize = t_iTh_y*t_iTh_z;
            int t_iSVDCache = m_iPairCols+1+1;//rv1[m_iPairCols]; scale; s

            size_t t_iSharedMemPerPairMat = sizeof(float) * (t_iPairMatSize + t_iWMatSize + t_iCorMatSize + t_iCacheYZSize + t_iSVDCache);
            int t_iPairMatsPerMultiProcessor = m_iSharedMemoryPerMultiProcessor/t_iSharedMemPerPairMat;
            std::cout << "Shared Memory Usage: " << t_iSharedMemPerPairMat << " Byte x " << t_iPairMatsPerMultiProcessor << std::endl;

            int t_iPairMatsPerBlock = ceil((float)(t_iPairMatsPerMultiProcessor)/(float)m_iMaxBlocksPerMultiProcessor);//=threadDim.x
/*DIRTY HACK*/  t_iPairMatsPerBlock = 2;//t_iPairMatsPerBlock > 2 ? 2 : t_iPairMatsPerBlock;//ToDo Debug when 3 Mats per Block we get the wrong result
            std::cout << "Pair Mats per Block: " << t_iPairMatsPerBlock << std::endl;

            size_t t_iSharedMemPerBlock = t_iSharedMemPerPairMat * t_iPairMatsPerBlock;

            blocks = dim3( /*7381*/ 64*m_iMultiProcessorCount, 1);
            threads = dim3( t_iPairMatsPerBlock, t_iTh_y, t_iTh_z);


            HANDLE_ERROR( hipEventRecord( start, 0 ) );

            PowellRapMusicSubcorr<<<blocks, threads, t_iSharedMemPerBlock>>>
                                              ( t_dev_pMatProj_LeadField->data(),
                                                t_dev_pMatProj_LeadField->rows(),
                                                t_dev_pMatProj_LeadField->cols(),
                                                m_dev_pPairIdxCombinations,
                                                t_dev_pRowIndezes,
                                                t_iNumVecElements,

                                                t_dev_pMatU_B->data(),
                                                t_iFullRank_U_B,
                                                t_dev_pRoh );
            HANDLE_ERROR( hipDeviceSynchronize() ); //to ensure that the kernel has completed

            HANDLE_ERROR( hipEventRecord( stop, 0 ) );
            HANDLE_ERROR( hipEventSynchronize( stop ) );
            HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                                start, stop ) );

            // free events
            HANDLE_ERROR( hipEventDestroy( start ) );
            HANDLE_ERROR( hipEventDestroy( stop ) );

            std::cout << "Time Elapsed: " << elapsedTime << " ms" << std::endl;
        //######## CUDA END ########



        //Find the maximum of correlation 
        //######## CUDA THRUST START ########
            //max_element returns an iterator, so to convert that into a position we subtract the iterator at the beginning of the vector.
            t_iMaxIdx = thrust::max_element(t_dev_vecRoh.begin(), t_dev_vecRoh.end()) - t_dev_vecRoh.begin(); 
            t_val_roh_k = t_dev_vecRoh[t_iMaxIdx];
        //######## THRUST CUDA END ########

            //Powell
            if(t_iMaxIdx == t_iMaxIdx_old)
            {
                t_iMaxFound = 1;
                break;
            }
            else
            {
                t_iMaxIdx_old = t_iMaxIdx;
                //get positions in sparsed leadfield from index combinations;
                t_iIdx1 = (*m_dev_pVecPairIdxCombinations)[2*t_iMaxIdx];
                t_iIdx2 = (*m_dev_pVecPairIdxCombinations)[2*t_iMaxIdx+1];
            }

            //set new index
            if(t_iIdx1 == t_iCurrentRow)
                t_iCurrentRow = t_iIdx2;
            else
                t_iCurrentRow = t_iIdx1;

        }

        // (Idx+1) because of MATLAB positions -> starting with 1 not with 0
        std::cout << "Iteration: " << r+1 << " of " << t_iMaxSearch
            << "; Correlation: " << t_val_roh_k<< "; Position (Idx+1): " << t_iIdx1+1 << " - " << t_iIdx2+1 <<"\n\n";

        //Calculations with the max correlated dipole pair G_k_1
        HPCMatrix<float> t_matG_k_1(t_matProj_LeadField.rows(),6);
        getLeadFieldPair(*m_pMatLeadField, t_matG_k_1, t_iIdx1, t_iIdx2);

        HPCMatrix<float> t_matProj_G_k_1(t_matOrthProj.rows(), t_matG_k_1.cols());
        t_matProj_G_k_1 = t_matOrthProj * t_matG_k_1;//Subtract the found sources from the current found source

        //Calculate source direction
        //source direction (p_pMatPhi) for current source r (phi_k_1)
        HPCMatrix<float> t_vec_phi_k_1(6, 1);
        subcorr(t_matProj_G_k_1, t_matU_B, t_vec_phi_k_1);//Correlate the current source to calculate the direction

        //Set return values
        p_pRapDipoles->insertSource(t_iIdx1, t_iIdx2, t_vec_phi_k_1.data(), t_val_roh_k);
        
        //Stop Searching when Correlation is smaller then the Threshold
        if (t_val_roh_k < m_dThreshold)
        {
            std::cout << "Searching stopped, last correlation " << t_val_roh_k;
            std::cout << " is smaller then the given threshold " << m_dThreshold << std::endl << std::endl;
            break;
        }
        
        //Calculate A_k_1 = [a_theta_1..a_theta_k_1] matrix for subtraction of found source
        calcA_k_1(t_matG_k_1, t_vec_phi_k_1, r, t_matA_k_1);

        //Calculate new orthogonal Projector (Pi_k_1)
        calcOrthProj(t_matA_k_1, t_matOrthProj);


        //#### CUDA START ####
            HANDLE_ERROR( hipMemcpy(   t_dev_pMatOrthProj->data(),
                                        t_matOrthProj.data(),
                                        sizeof(float) * t_matOrthProj.size(),
                                        hipMemcpyHostToDevice ) );
        //#### CUDA END ####


        //garbage collecting
            // free the memory allocated on the GPU
            delete t_dev_pMatU_B;

            // free the memory we allocated on the CPU

    }

    //garbage collecting
        // free the memory allocated on the GPU
        delete t_dev_pMatProj_LeadField;

        delete t_dev_pMatProj_Phi_s;

        delete t_dev_pMatOrthProj;

        delete t_dev_pMatPhi_s;

        // free the memory we allocated on the CPU

    std::cout << "##### Calculation of CUDA RAP MUSIC completed ######\n\n";

//     end = clock();
// 
//     float t_fElapsedTime = ( (float)(end-start) / (float)CLOCKS_PER_SEC ) * 1000.0f;
//     std::cout << "Time Elapsed: " << t_fElapsedTime << " ms" << std::endl << std::endl;


    //garbage collecting
    //ToDo


    return true;
}


//*************************************************************************************************************

int RapMusic_Cuda::nchoose2(int n)
{
    //nchoosek(n, k) with k = 2, equals n*(n-1)*0.5

    int t_iNumOfCombination = (int)(n*(n-1)*0.5);

    return t_iNumOfCombination;
}


//*************************************************************************************************************

//template <class T>
int  RapMusic_Cuda/*<T>*/::calcPhi_s(const HPCMatrix<float>& p_pMatMeasurement, cuHPCMatrix<float>* &p_dev_pMatPhi_s)
{
    //Calculate p_dev_pMatPhi_s
    HPCMatrix<float> t_matF;
    if (p_pMatMeasurement.cols() > p_pMatMeasurement.rows())
    {
        t_matF = makeSquareMat(p_pMatMeasurement); //FF^T
    }
    else
    {
        t_matF = p_pMatMeasurement;
    }

    SVD phi_sSVD(t_matF, 1);

    int t_r = getRank(phi_sSVD.singularValues());

    int t_iCols = t_r; //t_r < m_iN ? m_iN : t_r;

    if (p_dev_pMatPhi_s != NULL)
        delete p_dev_pMatPhi_s;

    //m_iNumChannels has to be equal to t_svdF.matrixU().rows()
    p_dev_pMatPhi_s = new cuHPCMatrix<float>(m_iNumChannels, t_iCols);

    //assign the signal subspace
    // copy the array from the CPU to the GPU
    HANDLE_ERROR(   hipMemcpy( p_dev_pMatPhi_s->data(), phi_sSVD.matrixU()->data(),
                                sizeof(float) * m_iNumChannels *t_iCols,
                                hipMemcpyHostToDevice ) );

    //ToDO Use jojos svd instead of cula

    //garbage collecting

    return t_r;
}


//*************************************************************************************************************
//Direction Subcorr
float RapMusic_Cuda::subcorr(HPCMatrix<float>& p_matProj_G, HPCMatrix<float>& p_matU_B, HPCMatrix<float>& p_vec_phi_k_1)
{
    //Orthogonalisierungstest wegen performance weggelassen -> ohne is es viel schneller
 
    SVD t_svdProj_G(p_matProj_G, 3);

    HPCMatrix<float> U_A_T(6, t_svdProj_G.matrixU()->rows());

    U_A_T = t_svdProj_G.matrixU()->transpose();

    HPCMatrix<float>* sigma_A = t_svdProj_G.singularValues();

    HPCMatrix<float>* V_A = t_svdProj_G.matrixV();

    //lt. Mosher 1998 ToDo: Only Retain those Components of U_A and U_B that correspond to nonzero singular values
    //for U_A and U_B the number of columns corresponds to their ranks
    //-> reduce to rank only when directions aren't calculated, otherwise use the full U_A_T

    HPCMatrix<float> Cor(6, p_matU_B.cols());

    //Step 2: compute the subspace correlation
    Cor = U_A_T*p_matU_B;//lt. Mosher 1998: C = U_A^T * U_B


    HPCMatrix<float>* t_vecSigma_C;

    //Step 4
    HPCMatrix<float>* U_C;

    if (Cor.cols() > Cor.rows())
    {
        Cor = Cor.transpose();//adjoint(); //for complex it has to be adjunct
        
        SVD svdOfCor_H(Cor, 2);

        U_C = new HPCMatrix<float>(svdOfCor_H.matrixV()->rows(), svdOfCor_H.matrixV()->cols());
        //because Cor Hermitesch U and V are exchanged
        memcpy(U_C->data(),svdOfCor_H.matrixV()->data(),(U_C->size()*sizeof(float)));

        t_vecSigma_C = new HPCMatrix<float>(svdOfCor_H.singularValues()->rows(), svdOfCor_H.singularValues()->cols());
        memcpy(t_vecSigma_C->data(),svdOfCor_H.singularValues()->data(),(t_vecSigma_C->size()*sizeof(float)));
    }
    else
    {
        SVD svdOfCor(Cor, 1);

        U_C = new HPCMatrix<float>(svdOfCor.matrixU()->rows(), svdOfCor.matrixU()->cols());
        memcpy(U_C->data(),svdOfCor.matrixU()->data(),(U_C->size()*sizeof(float)));

        t_vecSigma_C = new HPCMatrix<float>(svdOfCor.singularValues()->rows(), svdOfCor.singularValues()->cols());
        memcpy(t_vecSigma_C->data(),svdOfCor.singularValues()->data(),(t_vecSigma_C->size()*sizeof(float)));
    }

    //invert sigma A
    HPCMatrix<float> sigma_a_inv(sigma_A->rows(), sigma_A->rows());
    for (int i = 0; i < sigma_A->rows(); ++i)
    {
        sigma_a_inv(i,i) = 1/sigma_A->data()[i];
    }

    HPCMatrix<float> X(6,U_C->cols());
    X = ((*V_A)*sigma_a_inv)*(*U_C);//X = V_A*Sigma_A^-1*U_C

    float norm_X = 0;
    for(int i = 0; i < 6; ++i)
        norm_X += pow(X.data()[i], 2);

    norm_X = 1/sqrt(norm_X);

    //Multiply a scalar with an Array -> linear transform
    for(int i = 0; i < 6; ++i)
        p_vec_phi_k_1.data()[i] = X.data()[i]*norm_X;//u1 = x1/||x1|| this is the orientation

    //Step 3
    float ret_sigma_C;
    ret_sigma_C = t_vecSigma_C->data()[0]; //Take only the correlation of the first principal components

    //garbage collecting
    delete U_C;
    delete t_vecSigma_C;

    return ret_sigma_C;
}


//*************************************************************************************************************

void RapMusic_Cuda::calcA_k_1(  const HPCMatrix<float>& p_matG_k_1,
                                const HPCMatrix<float>& p_matPhi_k_1,
                                const int p_iIdxk_1,
                                HPCMatrix<float>& p_matA_k_1)
{
    //Calculate A_k_1 = [a_theta_1..a_theta_k_1] matrix for subtraction of found source
    HPCMatrix<float> t_vec_a_theta_k_1(p_matG_k_1.rows(),1);

    t_vec_a_theta_k_1 = p_matG_k_1*p_matPhi_k_1; // a_theta_k_1 = G_k_1*phi_k_1   this corresponds to the normalized signal component in subspace r

    memcpy( p_matA_k_1.data()+p_iIdxk_1*p_matA_k_1.rows(),
            t_vec_a_theta_k_1.data(),
            (p_matA_k_1.rows()*sizeof(float)));
}


//*************************************************************************************************************

void RapMusic_Cuda::calcOrthProj(const HPCMatrix<float>& p_matA_k_1, HPCMatrix<float>& p_matOrthProj)
{
    //Calculate OrthProj=I-A_k_1*(A_k_1'*A_k_1)^-1*A_k_1' //Wetterling -> A_k_1 = Gain

    HPCMatrix<float> t_matA_k_1_tmp(p_matA_k_1.cols(), p_matA_k_1.cols());
    t_matA_k_1_tmp = p_matA_k_1.transpose()/*adjoint()*/*p_matA_k_1;//A_k_1'*A_k_1 = A_k_1_tmp -> A_k_1' has to be adjoint for complex


    int t_size = t_matA_k_1_tmp.cols();

    while (!t_matA_k_1_tmp(t_size-1,t_size-1))
    {
        --t_size;
    }

    HPCMatrix<float> t_matA_k_1_tmp_inv(t_matA_k_1_tmp.rows(), t_matA_k_1_tmp.cols());

    HPCMatrix<float> t_matA_k_1_tmpsubmat = t_matA_k_1_tmp.get(0,0,t_size,t_size);
    LU t_matA_k_1_LU(&t_matA_k_1_tmpsubmat);
    for(int i = 0; i < t_matA_k_1_tmpsubmat.rows(); ++i)
        for(int j = 0; j < t_matA_k_1_tmpsubmat.cols(); ++j)
            t_matA_k_1_tmp_inv(i,j) = t_matA_k_1_LU.invert()(i,j);//(A_k_1_tmp)^-1 = A_k_1_tmp_inv


    t_matA_k_1_tmp.resize(p_matA_k_1.rows(), p_matA_k_1.cols());

    t_matA_k_1_tmp = p_matA_k_1*t_matA_k_1_tmp_inv;//(A_k_1*A_k_1_tmp_inv) = A_k_1_tmp


    HPCMatrix<float> t_matA_k_1_tmp2(p_matA_k_1.rows(), p_matA_k_1.rows());
    t_matA_k_1_tmp2 = t_matA_k_1_tmp*p_matA_k_1.transpose();//adjoint();//(A_k_1_tmp)*A_k_1' -> here A_k_1' is only transposed - it has to be adjoint

    HPCMatrix<float> I(m_iNumChannels,m_iNumChannels);
    I.diag(1.0);//setIdentity();

    p_matOrthProj = I-t_matA_k_1_tmp2; //OrthProj=I-A_k_1*(A_k_1'*A_k_1)^-1*A_k_1';

//     //garbage collecting
//     //ToDo
}


//*************************************************************************************************************
//ToDo don't make a real copy
void RapMusic_Cuda::getLeadFieldPair(   HPCMatrix<float>& p_matLeadField,
                                        HPCMatrix<float>& p_matLeadField_Pair,
                                        int p_iIdx1, int p_iIdx2)
{
    memcpy( p_matLeadField_Pair.data(),
            p_matLeadField.data()+p_iIdx1*3*p_matLeadField.rows(),
            (p_matLeadField.rows()*3*sizeof(float)));

    memcpy( p_matLeadField_Pair.data()+3*p_matLeadField.rows(),
            p_matLeadField.data()+p_iIdx2*3*p_matLeadField.rows(),
            (p_matLeadField.rows()*3*sizeof(float)));
}

}//Namespace